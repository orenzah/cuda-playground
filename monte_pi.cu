#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
__global__ void calculate(int seed, int *a , float* c, int N)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	hiprandState_t state;
	hiprand_init(seed + blockIdx.x, tid, 0, &state);
	//__shared__ int agg;
	//agg = agg + a[tid];
	int randX = hiprand(&state);
	float floatX = ((float)(randX % 1000000)) / 1000000;
	int randY = hiprand(&state);
	float floatY = ((float)(randY % 1000000)) / 1000000;
	if (powf(floatX,2) + powf(floatY,2) > 1)
	{
		a[tid] = 0;
	}
	else
	{
		a[tid] = 1;
	}
	//printf("tid: %d, random: %d\n", tid, rand % 25);
	__syncthreads();
	for (int j = 0; j < blockDim.x; j++)
	{
		int i = blockIdx.x * blockDim.x + j;
		c[blockIdx.x] = c[blockIdx.x] + ((float)a[i])/N;
	}
}
int main()
{
	float* gpu_x;
	int *gsum;
	int N = 1024;
	int B = 1 << 13;
	float* x = (float*) malloc(B * sizeof(float));

	hipMalloc((void**) & gpu_x, B*sizeof(float));
	hipMalloc((void**) & gsum, B*N*sizeof(int));
	hipMemcpy(gsum, &sum, 10*sizeof(int), hipMemcpyHostToDevice);
	calculate<<<B,N>>>(0, gsum, gpu_x, B*N);
	hipMemcpy(x, gpu_x, B*sizeof(int), hipMemcpyDeviceToHost);
	for (int i = 1; i < B; i++)
	{
		x[0] += x[i];
	}
	printf("Sum is %f\n", x[0]*4);
	hipFree(gpu_x);
	hipFree(gsum);
	free(x);
	return 0;
}
